#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/pad_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PadForward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int pad_l, const int pad_r,const int pad_t,const int pad_b) {
  CUDA_KERNEL_LOOP(index, count) {
    int i = index;  // Preserve the original value
    int height_out = height_in + pad_t + pad_b;
    int width_out = width_in + pad_l + pad_r;
    int w = i % width_in;
    i /= width_in;
    int h = i % height_in;
    i /= height_in;
    int c = i % channel;
    i /= channel;

    out[((i * channel + c) * height_out + h + pad_t) * width_out + pad_l + w] =
        in[index];
  }
}

template <typename Dtype>
__global__ void PadForwardPadZero(const int count, Dtype* out,
    const int num, const int channel, const int height_out, const int width_out,
    const int pad_l, const int pad_r,const int pad_t,const int pad_b) {
  CUDA_KERNEL_LOOP(index, count) {
    int w = index % width_out;
    int h = (index / width_out) % height_out;
    if (h < pad_t || h > height_out-1-pad_b || w < pad_l || w > width_out-1-pad_r) {
      out[index] = Dtype(0);
    }
  }
}

// No matching PadBackwardPadZero, since no gradient propagates
// through zero padding

template <typename Dtype>
void PadLayer<Dtype>::Forward_gpu(const std::vector<Blob<Dtype>*>& bottom,
    const std::vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int bcount = bottom[0]->count();
  const int tcount = top[0]->count();
  // First, set all data to be zero for the boundary pixels
  // CUDA_CHECK(hipMemset(top_data, 0, sizeof(Dtype) * tcount));
  // Copy the main body (not yet setting the padding)
  // NOLINT_NEXT_LINE(whitespace/operators)
  PadForward<Dtype><<<CAFFE_GET_BLOCKS(bcount), CAFFE_CUDA_NUM_THREADS>>>(
      bcount, bottom_data, top_data, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
      PAD_L_,PAD_R_,PAD_T_,PAD_B_);
  CUDA_POST_KERNEL_CHECK;

  // Padding
  switch (PAD_TYPE_) {
  case PadParameter::ZERO:
    // NOLINT_NEXT_LINE(whitespace/operators)
    PadForwardPadZero<Dtype><<<CAFFE_GET_BLOCKS(tcount),
                               CAFFE_CUDA_NUM_THREADS>>>(
        tcount, top_data, NUM_, CHANNEL_, HEIGHT_OUT_, WIDTH_OUT_,
        PAD_L_,PAD_R_,PAD_T_,PAD_B_);
    break;
  
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void PadBackward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int pad_l, const int pad_r,const int pad_t,const int pad_b) {
  CUDA_KERNEL_LOOP(index, count) {
    int i = index;  // Preserve original value
    int height_out = height_in + pad_t + pad_b;
    int width_out = width_in + pad_l + pad_r;
    int w = i % width_in;
    i /= width_in;
    int h = i % height_in;
    i /= height_in;
    int c = i % channel;
    i /= channel;
    out[index] = in[((i * channel + c) * height_out + h + pad_t) *
                    width_out + pad_l + w];
  }
}

template <typename Dtype>
void PadLayer<Dtype>::Backward_gpu(const std::vector<Blob<Dtype>*>& top,
    const std::vector<bool>& propagate_down,
    const std::vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* top_diff = top[0]->mutable_gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bcount = bottom[0]->count();
    const int tcount = top[0]->count();
    caffe_gpu_set(bcount, static_cast<Dtype>(0), bottom_diff);
    // In reverse order from Forward_gpu, so ...
    // Padding first. Operate within top to set the gradient in the
    // part to be copied to bottom.
    switch (PAD_TYPE_) {
    case PadParameter::ZERO:
      break;  // No gradient in the padding; it's constant
    }
    // Copy into place
    // NOLINT_NEXT_LINE(whitespace/operators)
    PadBackward<Dtype><<<CAFFE_GET_BLOCKS(bcount), CAFFE_CUDA_NUM_THREADS>>>(
        bcount, top_diff, bottom_diff, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
        PAD_L_,PAD_R_,PAD_T_,PAD_B_);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PadLayer);

}  // namespace caffe